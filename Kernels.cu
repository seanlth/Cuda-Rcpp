#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Kernels.h"

__global__ void logLikelihood2(double* a, double* b, double* data, unsigned int vector_length, unsigned int data_length, double* result)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    double sum = 0;
    for (unsigned int j = 0; j < data_length; j++) {
        sum += data[j];
    }
    
    result[i] = -b[i] * sum + data_length * (log(a[i]) + log(b[i])) - a[i] * (1 - exp(-b[i] * data[data_length-1]));
}

__global__ void logLikelihood(double* a, double* b, double* data, unsigned int vector_length, unsigned int data_length, double* result)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    double sum = 0;
    for (unsigned int j = 0; j < vector_length; j++) {
        sum += exp(a[j] + b[j]*data[i]);
    }
    result[i] = log(sum);
}

double* CUDALogLikelihood(double* data, unsigned int data_length, double** a, double** b, unsigned int vector_length, unsigned int iterations)
{   
    double* d_data; //device pointer for data
    unsigned int size_of_data = sizeof(double) * data_length;
    hipMalloc(&d_data, size_of_data);  //make space in GPU memory for data
    hipMemcpy(d_data, data, size_of_data, hipMemcpyHostToDevice); //copy data in a to GPU memory
    
    double* d_a; //device pointers for a and b
    double* d_b;
    unsigned int size_of_args = sizeof(double) * vector_length;
    hipMalloc(&d_a, size_of_args);  //make space in GPU memory for args
    hipMalloc(&d_b, size_of_args);  //make space in GPU memory for args

    double* tmp = new double[data_length];
    double* d_tmp;
    hipMalloc(&d_tmp, sizeof(double) * data_length);  //make space in GPU memory for result
    
    dim3 threadsPerBlock(32);
    
    int round_up = ( data_length + 32 - 1 ) / 32;
    
    dim3 blocksPerGrid( round_up );

    double* result = new double[iterations];
    std::fill(result, result+iterations, 0);
    
    for (unsigned int i = 0; i < iterations; i++) {
        
        double* a_temp = a[i];
        double* b_temp = b[i];
        
        hipMemcpy(d_a, a_temp, size_of_args, hipMemcpyHostToDevice); //copy 'a' to GPU memory
        hipMemcpy(d_b, b_temp, size_of_args, hipMemcpyHostToDevice); //copy 'b' to GPU memory

        logLikelihood<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_data, vector_length, data_length, d_tmp);
        
        hipMemcpy(tmp, d_tmp, sizeof(double) * data_length, hipMemcpyDeviceToHost);

        //replace for CUDA horizontal sum
        for (unsigned int j = 0; j < data_length; j++) {
            result[i] += tmp[j];
        }
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_data);
    hipFree(d_tmp);
    
    hipDeviceReset();
    
    return result;
}


